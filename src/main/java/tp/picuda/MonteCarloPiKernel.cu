
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

extern "C"
__global__ void monteCarloPi(double *count, int totalIterations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // linearisation de tableau

    if (idx >= totalIterations) return;

    hiprandState state;
    hiprand_init(idx, 0, 0, &state);

    double x = hiprand_uniform(&state);
    double y = hiprand_uniform(&state);

    if (x * x + y * y <= 1.0) {
        count[idx] = 1.0;  // Stocke les hits de chaque thread dans le tableau
    } else {
        count[idx] = 0.0;
    }
}
